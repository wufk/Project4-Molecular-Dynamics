#include "hip/hip_runtime.h"
#include "md.h"

#include <cstdio>
#include <cmath>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>

#include "hip/device_functions.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

#include "memory.h"
#include "random.h"

float m = 1; float inv_m = 1. / m;

float dt = 0.0005;
float hdt = 0.5 * dt;

float sigma = 1, epsilon = 1, A, B, C, D;
float dcut = 2.5 * sigma;
float dcut2 = dcut * dcut;

float t = 0.0;

float tau = 0.01 / dt;

float kB = 1., ke = 0.0, pe = 0.0;
float T;
float T0 = 0.5, Tt = 10.5;

int Crate = 20;

int ifreq = 10, nstep = 100;
int N = (Tt - T0) / (Crate * dt * nstep);

int nunit = 4;
int nall = 4;

char Atom[3] = "Cr";

Memory* M = new Memory();;
RanPark* rnd = new RanPark(1234);;

float ncell[3] = { 4.0f, 4.0f, 4.0f };
float a0[3] = { 1.5f, 1.5f, 1.5f };
glm::vec3 L(ncell[0] * a0[0], ncell[1] * a0[1], ncell[2] * a0[2]);
glm::vec3 hL(L[0] / 2, L[1] / 2, L[2] / 2);

glm::vec3 *dev_vel = nullptr;
glm::vec3 *dev_pos = nullptr;
glm::vec3 *dev_force = nullptr;
glm::vec3 *h_vel = nullptr;
glm::vec3 *h_pos = nullptr;
glm::vec3 *h_force = nullptr;
float *ke_idata = nullptr;
float *ke_odata = nullptr;
float *dev_pe = nullptr;
const int threads = 256;

thrust::minstd_rand rng;
thrust::uniform_real_distribution<float> unitDistrib(-0.5, 0.5);

hipEvent_t start;
hipEvent_t end;

hiprandState_t* states = nullptr;

////////////////////////////////////////////// Velocity //////////////////////////////////////////////
__global__ void kernInitVel(int n, glm::vec3 *vel, hiprandState_t* states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;

	float vx = hiprand_uniform(&(states[tid])) - 0.5f;
	float vy = hiprand_uniform(&(states[tid])) - 0.5f;
	float vz = hiprand_uniform(&(states[tid])) - 0.5f;
	vel[tid] = glm::vec3(vx,vy,vz);
}

__global__ void kernVelMinus(int n, glm::vec3 *vel, glm::vec3 mon) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;

	vel[tid] -= mon;
}

__global__ void kernVelMultiply(int n, glm::vec3 *vel, float gama) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;

	vel[tid] *= gama;
}

__global__ void kernComputeDotProduct(int n, glm::vec3 *vel, float *d_odata)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;

	d_odata[tid] = glm::dot(vel[tid], vel[tid]);
}

//////////////////////////////////// reduce energy //////////////////////////////////////////////////////////
__global__ void reduce_energy(const float* d_idata, float* d_odata, int n)
{
	extern __shared__ float shm[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
		shm[threadIdx.x] = d_idata[idx];

	__syncthreads();

	for (int c = blockDim.x / 2; c > 0; c >>= 1)
	{

		if (threadIdx.x < c)
			shm[threadIdx.x] += shm[threadIdx.x + c];

		__syncthreads();
	}

	if (threadIdx.x == 0)
		d_odata[blockIdx.x] = shm[0];
}

float reduce_energy_wrapper(const float *d_idata, float *d_odata, const int elements)
{
	int dimThreads = threads;
	int dimBlocks = (elements + dimThreads - 1) / (dimThreads);

	if (elements < dimThreads) {
		float *h_blocks = (float *)malloc(elements * sizeof(float));
		hipMemcpy(h_blocks, d_odata, elements * sizeof(float), hipMemcpyDeviceToHost);

		float gpu_result = 0;

		for (int i = 0; i < elements; i++)
			gpu_result += h_blocks[i];

		free(h_blocks);

		return gpu_result;
	}
	else {
		reduce_energy << <dimBlocks, dimThreads, sizeof(float) * dimThreads >> >(d_idata, d_odata, elements);

		return reduce_energy_wrapper(d_odata, d_odata, dimBlocks);
	}
}

/////////////////////////////////////////// Reduce velocity //////////////////////////////////////////////
__global__ void vel_reduce(const glm::vec3* d_idata, glm::vec3* d_odata, int n)
{
	extern __shared__ glm::vec3 smem[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
		smem[threadIdx.x] = d_idata[idx];

	__syncthreads();

	for (int c = blockDim.x / 2; c > 0; c >>= 1)
	{
		if (threadIdx.x < c)
			smem[threadIdx.x] += smem[threadIdx.x + c];

		__syncthreads();
	}

	if (threadIdx.x == 0)
		d_odata[blockIdx.x] = smem[0];
}

glm::vec3 vel_reduce_wrapper(const glm::vec3 *d_idata, glm::vec3 *d_odata, const int elements)
{
	int dimThreads = threads;
	int dimBlocks = (elements + dimThreads - 1) / (dimThreads);

	if (elements < dimThreads) {
		glm::vec3 *h_blocks = (glm::vec3 *)malloc(elements * sizeof(glm::vec3));
		hipMemcpy(h_blocks, d_odata, elements * sizeof(glm::vec3), hipMemcpyDeviceToHost);

		glm::vec3 gpu_result = glm::vec3(0.0f, 0.0f, 0.0f);

		for (int i = 0; i < elements; i++)
			gpu_result += h_blocks[i];

		free(h_blocks);

		return gpu_result;
	}
	else {
		vel_reduce << <dimBlocks, dimThreads, sizeof(glm::vec3) * dimThreads >> >(d_idata, d_odata, elements);

		return vel_reduce_wrapper(d_odata, d_odata, dimBlocks);
	}
}

////////////////////////////////////////////////////
__global__ void kernNaiveVelocityIntegration(int n, glm::vec3 *vel, glm::vec3 *force, float inv_m, float hdt)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;
	vel[tid] += force[tid] * inv_m * hdt;
}

__global__ void kernNaivePositionIntegration(int n, glm::vec3 *pos, glm::vec3 *vel, float dt)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;
	pos[tid] += dt * vel[tid];
}

__global__ void kernNaiveForce(int n, glm::vec3 *pos, glm::vec3 *force, glm::vec3 *vel, float coef, glm::vec3 hL, glm::vec3 L,
	float dcut2, float A, float B, float C, float D, float m, float tau, float *pe, hiprandState_t* states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;

	float wx = hiprand_uniform(&(states[tid])) - 0.5f;
	float wy = hiprand_uniform(&(states[tid])) - 0.5f;
	float wz = hiprand_uniform(&(states[tid])) - 0.5f;
	glm::vec3 w = glm::vec3(wx,wy,wz);

	for (int i = tid + 1; i < n; i++) {

		glm::vec3 dx = pos[i] - pos[tid];
		while (dx.x > hL.x) dx.x -= L.x;
		while (dx.y > hL.y) dx.y -= L.y;
		while (dx.z > hL.z) dx.z -= L.z;
		while (-dx.x > hL.x) dx.x += L.x;
		while (-dx.y > hL.y) dx.y += L.y;
		while (-dx.z > hL.z) dx.z += L.z;

		float r2 = glm::dot(dx, dx);
		float r6 = r2 * r2 *r2;
		float r12 = r6 * r6;

		if (r2 < dcut2) {
			dx *= (A * 1. / r12 + B * 1. / r6) / r2;
			atomicAdd(&(force[tid].x), -dx.x);
			atomicAdd(&(force[tid].y), -dx.y);
			atomicAdd(&(force[tid].z), -dx.z);
			atomicAdd(&(force[i].x), dx.x);
			atomicAdd(&(force[i].y), dx.y);
			atomicAdd(&(force[i].z), dx.z);
			atomicAdd(pe, C * 1. / r12 + D * 1. / r6);
		}
	}
	__syncthreads();
	force[tid] += -m * tau * vel[tid] + coef * w;
}

void forceCPU(int k) {
	float TT = T0;
	float Gc = (Tt - T0) / (N - 1);
	TT = TT + k * Gc;
	//printf("TT: %f\n", TT);
	pe = 0.0;
	float coef = sqrt(24. * tau * m * kB * TT / dt);//coef to calculate w
	float dcut = 2.5 * sigma;
	float dcut2 = dcut * dcut;

	for (int i = 0; i < nall; i++){
		h_force[i] = glm::vec3(0.0f, 0.0f, 0.0f);
	}

	for (int i = 0; i < nall - 1; i++) {
		for (int j = i + 1; j < nall; j++) {
			glm::vec3 dx = h_pos[j] - h_pos[i];
			for (int k = 0; k < 3; k++) {
				while (dx.x > hL.x) dx.x -= L.x;
				while (dx.y > hL.y) dx.y -= L.y;
				while (dx.z > hL.z) dx.z -= L.z;
				while (dx.x < -hL.x) dx.x += L.x;
				while (dx.y < -hL.y) dx.y += L.y;
				while (dx.z < -hL.z) dx.z += L.z;
			}
			float r2 = glm::dot(dx, dx);

			float r6 = r2 * r2 *r2;
			float r12 = r6 * r6;

			if (r2 < dcut2) {
				dx *= (A * 1. / r12 + B * 1. / r6) / r2;
				h_force[i] -= dx;
				h_force[j] += dx;
				pe += C * 1. / r12 + D * 1. / r6;
			}
		}
	}

	for (int i = 0; i < nall; i++) {
		float a = (float)unitDistrib(rng);
		float b = (float)unitDistrib(rng);
		float c = (float)unitDistrib(rng);
		glm::vec3 w(a, b, c);
		w *= coef;
		h_force[i] += -m * tau * h_vel[i] + w;
	}

}

__global__ void kernInitRandom(int n, unsigned int seed, hiprandState_t* states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) return;
	hiprand_init(seed, tid, 0, &states[tid]);
}

void MD::MD_init()
{
	//init variables
	float sigma3 = sigma  * sigma * sigma;
	float sigma6 = sigma3 * sigma3;
	float sigma12 = sigma6 * sigma6;
	A = 48. * epsilon * sigma12;
	B = -24. * epsilon * sigma6;
	C = 4. * epsilon * sigma12;
	D = -4. * epsilon * sigma6;

	for (int i = 0; i < 3; i++) nall *= ncell[i];

	int dimThreads = threads;
	int dimBlocks = (nall + dimThreads - 1) / (dimThreads);

	hipMalloc((void**)&dev_pe, sizeof(float));
	hipMalloc((void**)&dev_pos, nall * sizeof(glm::vec3));
	hipMalloc((void**)&dev_vel, nall * sizeof(glm::vec3));
	hipMalloc((void**)&dev_force, nall * sizeof(glm::vec3));
	hipMalloc((void**)&ke_idata, nall * sizeof(glm::vec3));
	hipMalloc((void**)&ke_odata, dimBlocks * sizeof(glm::vec3));
	hipMalloc((void**)&states, nall * sizeof(hiprandState_t));

	kernInitRandom << <dimBlocks, dimThreads >> > (nall, 0, states);


	h_pos = new glm::vec3[nall];
	h_vel = new glm::vec3[nall];
	h_force = new glm::vec3[nall];

	//init cell and pos
	float **cell = nullptr;
	M->create(cell, 4, 3, "cell");
	cell[0][0] = cell[0][1] = cell[0][2] = 0.0;
	cell[1][0] = 0.0; cell[1][1] = 0.5 * a0[1]; cell[1][2] = 0.5 * a0[2];
	cell[2][0] = 0.5 * a0[0]; cell[2][1] = 0.0; cell[2][2] = 0.5 * a0[2];
	cell[3][0] = 0.5 * a0[0]; cell[3][1] = 0.5 * a0[1]; cell[3][2] = 0.0;
	int ii = 0;
	for (int ix = 0; ix < ncell[0]; ix++) {
		for (int iy = 0; iy < ncell[1]; iy++) {
			for (int iz = 0; iz < ncell[2]; iz++) {
				for (int iu = 0; iu < nunit; iu++) {
					h_pos[ii].x = float(ix) * a0[0] + cell[iu][0];
					h_pos[ii].y = float(iy) * a0[1] + cell[iu][1];
					h_pos[ii].z = float(iz) * a0[2] + cell[iu][2];
					++ii;
				}
			}
		}
	}
	hipMemcpy(dev_pos, h_pos, nall * sizeof(glm::vec3), hipMemcpyHostToDevice);
	M->destroy(cell);

	//////////////////////////////
	kernInitVel << <dimBlocks, dimThreads >> > (nall, dev_vel, states);
	glm::vec3 *vel_odata = nullptr;
	hipMalloc((void**)&vel_odata, dimBlocks * sizeof(glm::vec3));
	glm::vec3 mon = vel_reduce_wrapper(dev_vel, vel_odata, nall);
	hipFree(vel_odata);
	mon /= float(nall);
	kernVelMinus << <dimBlocks, dimThreads >> >(nall, dev_vel, mon);

	kernComputeDotProduct << <dimBlocks, dimThreads >> > (nall, dev_vel, ke_idata);
	ke = reduce_energy_wrapper(ke_idata, ke_odata, nall);

	ke *= 0.5 * m; T = ke / (1.5 * float(nall) * kB);
	float gamma = sqrt(T0 / T);

	ke = 0.0;
	kernVelMultiply << <dimBlocks, dimThreads >> > (nall, dev_vel, gamma);
	kernComputeDotProduct << <dimBlocks, dimThreads >> > (nall, dev_vel, ke_idata);
	ke = reduce_energy_wrapper(ke_idata, ke_odata, nall);

	ke *= 0.5 * m;
	T = ke / (1.5 * float(nall) * kB);

	hipMemset(dev_force, 0, sizeof(glm::vec3) * nall);
	hipMemset(dev_pe, 0, sizeof(float));

	float coef = sqrt(24. * tau * m * kB * T0 / dt);
	kernNaiveForce << <dimBlocks, dimThreads >> >(nall, dev_pos, dev_force, dev_vel, coef, hL, L, dcut2, A, B, C, D, m, tau, dev_pe, states);
	hipMemcpy(&pe, dev_pe, sizeof(float), hipMemcpyDeviceToHost);
	//////////////////////////////


	//init velocity
	//glm::vec3 mon(0.0f, 0.0f, 0.0f);
	//for (int i = 0; i < nall; i++) {

	//	float a = (float)unitDistrib(rng);
	//	float b = (float)unitDistrib(rng);
	//	float c = (float)unitDistrib(rng);
	//	h_vel[i] = glm::vec3(a,b,c);
	//	mon += h_vel[i];
	//}
	//mon /= nall; ke = 0.0f;
	//for (int i = 0; i < nall; i++) {
	//	h_vel[i] -= mon;
	//	ke += glm::dot(h_vel[i], h_vel[i]);
	//}
	//ke *= 0.5 * m;
	//T = ke / (1.5 * float(nall) * kB);
	//float gama = sqrt(T0 / T);

	//ke = 0.0f;
	//for (int i = 0; i < nall; i++) {
	//	h_vel[i] *= gama;
	//	ke += glm::dot(h_vel[i], h_vel[i]);
	//}

	//ke *= 0.5 * m;
	//T = ke / (1.5 * float(nall) * kB);

	//forceCPU(0);

	printf("ke = %f pe = %f T = %f\n", ke, pe, T);
}

void MD::MD_free()
{
	hipFree(dev_force);
	hipFree(dev_pe);
	hipFree(dev_vel);
	hipFree(dev_pos);
	hipFree(states);
	hipFree(ke_idata);
	hipFree(ke_odata);

	delete[] h_force;
	delete[] h_pos;
	delete[] h_vel;

	delete M;
	delete rnd;
}

void MD::MD_run()
{
	for (int i = 1; i < N; i++) {
		MD_Loop(i);
	}
}

__global__ void testinitrandom(int n, int seed, hiprandState *states) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void kernTestRandom(int n, hiprandState *states, glm::vec3 *vec) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < n) {
		float a = hiprand_uniform(&states[idx]) - 0.5;
		float b = hiprand_uniform(&states[idx]) - 0.5;
		float c = hiprand_uniform(&states[idx]) - 0.5;
		vec[idx] = glm::vec3(a, b, c);
	}

}

void MD::MD_Loop(int k)
{
	int dimThreads = threads;
	int dimBlocks = (nall + dimThreads - 1) / (dimThreads);

	for (int i = 0; i < nstep; i++) {
		kernNaiveVelocityIntegration << <dimBlocks, dimThreads >> > (nall, dev_vel, dev_force, inv_m, hdt);

		kernNaivePositionIntegration << <dimBlocks, dimThreads >> > (nall, dev_pos, dev_vel, dt);

		hipMemset(dev_force, 0.0f, sizeof(glm::vec3) * nall);
		hipMemset(dev_pe, 0.0f, sizeof(float));
		double TT = T0 + (Tt - T0) / (N - 1) * k;
		float coef = sqrt(24. * tau * m * kB * TT / dt);
		kernNaiveForce << <dimBlocks, dimThreads >> >(nall, dev_pos, dev_force, dev_vel, coef, hL, L, dcut2, A, B, C, D, m, tau, dev_pe, states);

		kernNaiveVelocityIntegration << <dimBlocks, dimThreads >> > (nall, dev_vel, dev_force, inv_m, hdt);

		if (i % ifreq == 0) {
			kernComputeDotProduct << <dimBlocks, dimThreads >> > (nall, dev_vel, ke_idata);
			ke = reduce_energy_wrapper(ke_idata, ke_odata, nall);

			ke *= 0.5 * m; t += dt;
			T = ke / (1.5 * float(nall) * kB);

			hipMemcpy(&pe, dev_pe, sizeof(float), hipMemcpyDeviceToHost);
			printf("step %d ke %f pe %f T %f TT %f coef %f\n", i, ke, pe, T, TT, coef);
		}

	}
	//for (int i = 0; i < nstep; i++) {

	//	for (int j = 0; j < nall; j++) {
	//		h_vel[j] += h_force[j] * hdt;
	//	}

	//	for (int j = 0; j < nall; j++) {
	//		h_pos[j] += h_vel[j] * hdt;
	//	}

	//	float TT = T0;
	//	float Gc = (Tt - T0) / (N - 1);
	//	TT = TT + k * Gc;
	//	pe = 0.0;
	//	float coef = sqrt(24. * tau * m * kB * TT / dt);//coef to calculate w
	//	forceCPU(k);

	//	for (int j = 0; j < nall; j++) {
	//		h_vel[j] += h_force[j] * hdt;
	//	}

	//	ke = 0.0f;
	//	for (int i = 0; i < nall; i++) {
	//		ke += glm::dot(h_vel[i], h_vel[i]);
	//	}

	//	ke *= 0.5 * m;
	//	T = ke / (1.5 * float(nall) * kB);

	//	if (i % ifreq == 0) printf("step %d ke %f pe %f T %f TT %f coef %f\n", i, ke, pe, T, TT, coef);
	//}
}
